#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <stdlib.h>

#define TILE 16

__global__ void gemmTiled(float* da, float* db, float* dc, int N){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ float As[TILE][TILE];
    __shared__ float Bs[TILE][TILE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float sum = 0.0f;
    for (int t = 0; t < (N + TILE - 1) / TILE; ++t) {
      As[ty][tx] = (row < N && t*TILE + tx < N)
        ? da[row * N + (t*TILE + tx)]
        : 0.0f;

      Bs[ty][tx] = (col < N && t*TILE + ty < N)
        ? db[(t*TILE + ty) * N + col]
        : 0.0f;

      __syncthreads();

      for (int k = 0; k < TILE; ++k)
        sum += As[ty][k] * Bs[k][tx];

      __syncthreads();
    }

    if (row < N && col < N){
      dc[row * N + col] = sum;
    }

}

int main(){
    printf(" N      tiledTime(ms)  tiledGFLOPs   tiledGFLOPs/s   cuBLASTime(ms)  cuBLASGFLOPs   cuBLASGFLOPs/s\n");
    for(int N = 512; N <= 8192; N += 512){
        size_t size = N * N * sizeof(float);
        float* a = (float*)malloc(size);
        float* b = (float*)malloc(size);
        float* c = (float*)malloc(size);
        float* c_cublas = (float*)malloc(size);

        for(int i=0;i<N;i++){
            for(int j=0;j<N;j++){
                a[i*N+j] = i+j;
                b[i*N+j] = 2*(i+j);
            }
        }

        float *da, *db, *dc, *dc_cublas;
        hipMalloc((void**)&da, size);
        hipMalloc((void**)&db, size);
        hipMalloc((void**)&dc, size);
        hipMalloc((void**)&dc_cublas, size);

        hipMemcpy(da, a, size, hipMemcpyHostToDevice);
        hipMemcpy(db, b, size, hipMemcpyHostToDevice);

        dim3 tpb(TILE, TILE, 1);
        dim3 bpg((N+TILE-1)/TILE, (N+TILE-1)/TILE, 1);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        gemmTiled<<<bpg, tpb>>>(da, db, dc, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float naiveTimeMs = 0.0f;
        hipEventElapsedTime(&naiveTimeMs, start, stop);

        float naiveGFLOPs = 2.0f * N * N * N / 1e9f;
        float naiveGFLOPsPerSec = naiveGFLOPs / (naiveTimeMs / 1000.0f);

        hipblasHandle_t handle;
        hipblasCreate(&handle);
        float alpha = 1.0f;
        float beta = 0.0f;

        hipEventRecord(start);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, db, N, da, N, &beta, dc_cublas, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float cublasTimeMs = 0.0f;
        hipEventElapsedTime(&cublasTimeMs, start, stop);

        float cublasGFLOPs = 2.0f * N * N * N / 1e9f;
        float cublasGFLOPsPerSec = cublasGFLOPs / (cublasTimeMs / 1000.0f);

        printf("%4d     %10.3f     %10.2f     %13.2f     %13.3f     %12.2f     %15.2f\n",
               N, naiveTimeMs, naiveGFLOPs, naiveGFLOPsPerSec,
               cublasTimeMs, cublasGFLOPs, cublasGFLOPsPerSec);

        hipblasDestroy(handle);
        free(a); free(b); free(c); free(c_cublas);
        hipFree(da); hipFree(db); hipFree(dc); hipFree(dc_cublas);
        hipEventDestroy(start); hipEventDestroy(stop);
    }
    return 0;
}
