#include <hip/hip_runtime.h>
#include <stdio.h> 

#define TIME_KERNEL(...) do { \
    hipEvent_t start, stop; \
    hipEventCreate(&start); \
    hipEventCreate(&stop); \
    hipEventRecord(start); \
    __VA_ARGS__; \
    hipEventRecord(stop); \
    hipEventSynchronize(stop); \
    float ms = 0.0f; \
    hipEventElapsedTime(&ms, start, stop); \
    printf("Time taken by %s: %.3f ms\n", #__VA_ARGS__, ms); \
    hipEventDestroy(start); \
    hipEventDestroy(stop); \
} while(0)



__global__ void atomicAddNaive(const double* d_a, double* d_out, int N){
  int idx = blockDim.x * blockIdx.x  + threadIdx.x;
  if (idx < N){
    atomicAdd(d_out, d_a[idx]);
  }
}


int main(){
  int N = 1 << 20;
  int size = N * sizeof(double);
  double* a = (double*) malloc(size);
  double* out = (double*) malloc(sizeof(double));

  *out = 0.0;
  for(int i=0; i < N; i++) a[i] = i;

  double* d_out;
  double* d_a;
  hipMalloc((void**)&d_a, size);
  hipMalloc((void**)&d_out, sizeof(double));
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_out, out, sizeof(double), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(256);
  dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);
  TIME_KERNEL(atomicAddNaive<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_out, N));
  hipDeviceSynchronize();

  hipMemcpy(out, d_out, sizeof(double), hipMemcpyDeviceToHost);

  printf("Result: %lf\n", *out);

  free(a);
  free(out);
  hipFree(d_a);
  hipFree(d_out);

  return 0;
}
