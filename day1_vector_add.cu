#include <hip/hip_runtime.h>
#include <stdio.h>

#define TIME_KERNEL(...) do { \
    hipEvent_t start, stop; \
    hipEventCreate(&start); \
    hipEventCreate(&stop); \
    hipEventRecord(start); \
    __VA_ARGS__; \
    hipEventRecord(stop); \
    hipEventSynchronize(stop); \
    float ms = 0.0f; \
    hipEventElapsedTime(&ms, start, stop); \
    printf("Time taken by %s: %.3f ms\n", #__VA_ARGS__, ms); \
    hipEventDestroy(start); \
    hipEventDestroy(stop); \
} while(0)



__global__ void vectorAdd(float* a, float* b, float* c, int N){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N){
    c[idx] = b[idx] + a[idx];
  }
}

int main(){
  int N = 1 << 20;
  int size = N * sizeof(float);

  float* a = (float*) malloc(size);
  float* b = (float*) malloc(size);
  float* c = (float*) malloc(size);

  for(int i=0; i < N; i++){
    a[i] = i;
  }

  for(int i=0; i < N; i++){
    b[i] = 2*i;
  }
 
  float* da;
  float* db;
  float* dc;

  hipMalloc((void**)&da, size);
  hipMalloc((void**)&db, size);
  hipMalloc((void**)&dc, size);

  hipMemcpy(da, a, size, hipMemcpyHostToDevice);
  hipMemcpy(db, b, size, hipMemcpyHostToDevice);


  dim3 threadsPerBlock(256, 1, 1);
  dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

  TIME_KERNEL(vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, N));
  hipDeviceSynchronize();

  hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);

  for(int i=0; i < 5; i++){
    printf("%f, ", c[i]);
  }

  hipFree(da);
  hipFree(db);
  hipFree(dc);

  free(a);
  free(b);
  free(c);

  return 0;
}

