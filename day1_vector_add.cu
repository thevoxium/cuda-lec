#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(float* a, float* b, float* c, int N){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N){
    c[idx] = b[idx] + a[idx];
  }
}

int main(){
  int N = 1 << 20;
  int size = N * sizeof(float);

  float* a = (float*) malloc(size);
  float* b = (float*) malloc(size);
  float* c = (float*) malloc(size);

  for(int i=0; i < N; i++){
    a[i] = i;
  }

  for(int i=0; i < N; i++){
    b[i] = 2*i;
  }
 
  float* da;
  float* db;
  float* dc;

  hipMalloc((void**)&da, size);
  hipMalloc((void**)&db, size);
  hipMalloc((void**)&dc, size);

  hipMemcpy(da, a, size, hipMemcpyHostToDevice);
  hipMemcpy(db, b, size, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  dim3 threadsPerBlock(256, 1, 1);
  dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

  hipEventRecord(start);
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, N);
  hipDeviceSynchronize();
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);
  printf("Kernel execution time: %f ms\n", elapsed);


  hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);

  for(int i=0; i < 5; i++){
    printf("%f, ", c[i]);
  }

  hipFree(da);
  hipFree(db);
  hipFree(dc);

  free(a);
  free(b);
  free(c);

  return 0;
}

