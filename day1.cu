#include <hip/hip_runtime.h>

__global__ void hello(){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  printf("Block Id: %d, Thread Id: %d", blockIdx.x, threadIdx.x);
}

int main(){
  int N = 1 << 10;
  dim3 threadsPerBlock(256, 1, 1);
  dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);
  hello<<<blocksPerGrid, threadsPerBlock>>>();
hipDeviceSynchronize();
  return 0;
}
