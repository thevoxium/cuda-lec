
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <stdlib.h>

#define __threadCount 16

__global__ void gemmNaive(float* da, float* db, float* dc, int N){
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < N && j < N){
        float sum = 0.0f;
        for (int k=0; k < N; ++k){
            sum += da[i * N + k] * db[k * N + j];
        }
        dc[i * N + j] = sum;
    }
}

float timeKernel(void(*kernel)(float*, float*, float*, int), float* da, float* db, float* dc, int N, dim3 bpg, dim3 tpb){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel<<<bpg, tpb>>>(da, db, dc, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return ms;
}

int main(){
    printf(" N      NaiveTime(ms)  NaiveGFLOPs  cuBLASTime(ms)  cuBLASGFLOPs\n");
    for(int N = 512; N <= 8192; N += 512){
        size_t size = N * N * sizeof(float);
        float* a = (float*)malloc(size);
        float* b = (float*)malloc(size);
        float* c = (float*)malloc(size);
        float* c_cublas = (float*)malloc(size);

        for(int i=0;i<N;i++)
            for(int j=0;j<N;j++){
                a[i*N+j] = i+j;
                b[i*N+j] = 2*(i+j);
            }

        float *da, *db, *dc, *dc_cublas;
        hipMalloc((void**)&da, size);
        hipMalloc((void**)&db, size);
        hipMalloc((void**)&dc, size);
        hipMalloc((void**)&dc_cublas, size);

        hipMemcpy(da, a, size, hipMemcpyHostToDevice);
        hipMemcpy(db, b, size, hipMemcpyHostToDevice);

        dim3 tpb(__threadCount, __threadCount, 1);
        dim3 bpg((N+__threadCount-1)/__threadCount, (N+__threadCount-1)/__threadCount, 1);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        gemmNaive<<<bpg, tpb>>>(da, db, dc, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float naiveTime = 0.0f;
        hipEventElapsedTime(&naiveTime, start, stop);

        float gflops_naive = 2.0f * N * N * N / (naiveTime * 1e6f);

        hipblasHandle_t handle;
        hipblasCreate(&handle);
        float alpha = 1.0f;
        float beta = 0.0f;

        hipEventRecord(start);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, db, N, da, N, &beta, dc_cublas, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float cublasTime = 0.0f;
        hipEventElapsedTime(&cublasTime, start, stop);

        float gflops_cublas = 2.0f * N * N * N / (cublasTime * 1e6f);

        printf("%4d     %10.3f     %10.2f     %10.3f     %10.2f\n", N, naiveTime, gflops_naive, cublasTime, gflops_cublas);

        hipblasDestroy(handle);
        free(a); free(b); free(c); free(c_cublas);
        hipFree(da); hipFree(db); hipFree(dc); hipFree(dc_cublas);
        hipEventDestroy(start); hipEventDestroy(stop);
    }
    return 0;
}
