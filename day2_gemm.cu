#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <stdlib.h>

#define TIME_KERNEL(...) do { \
    hipEvent_t start, stop; \
    hipEventCreate(&start); \
    hipEventCreate(&stop); \
    hipEventRecord(start); \
    __VA_ARGS__; \
    hipEventRecord(stop); \
    hipEventSynchronize(stop); \
    float ms = 0.0f; \
    hipEventElapsedTime(&ms, start, stop); \
    printf("Time taken by %s: %.3f ms\n", #__VA_ARGS__, ms); \
    hipEventDestroy(start); \
    hipEventDestroy(stop); \
} while(0)

#define N (1 << 13)
#define __threadCount 16

__global__ void gemmNaive(float* da, float* db, float* dc){
  int j = blockDim.x * blockIdx.x + threadIdx.x;
  int i = blockDim.y * blockIdx.y + threadIdx.y;
  if(i < N && j < N){
    float sum = 0.0f;
    for (int k=0; k < N; ++k){
      sum += (da[i * N + k] * db[k * N + j]);
    }
    dc[i * N + j] = sum;
  }
}

int main(){
    size_t size = N * N * sizeof(float);
    float* a = (float*)malloc(size);
    float* b = (float*)malloc(size);
    float* c = (float*)malloc(size);
    float* c_cublas = (float*)malloc(size);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            a[i * N + j] = i + j;
            b[i * N + j] = 2 * (i + j);
        }
    }

    float* da;
    float* db;
    float* dc;
    float* dc_cublas;

    hipMalloc((void**)&da, size);
    hipMalloc((void**)&db, size);
    hipMalloc((void**)&dc, size);
    hipMalloc((void**)&dc_cublas, size);

    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size, hipMemcpyHostToDevice);

    dim3 tpb(__threadCount, __threadCount, 1);
    dim3 bpg((N+__threadCount-1)/__threadCount, (N+__threadCount-1)/__threadCount, 1);

    TIME_KERNEL(gemmNaive<<<bpg, tpb>>>(da, db, dc));
    hipDeviceSynchronize();

    hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);

    //cublas implementation

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f;
    float beta = 0.0f;

    TIME_KERNEL(hipblasSgemm(handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N,
        N,
        N,
        &alpha,
        db, N,
        da, N,
        &beta,
        dc_cublas, N));

    hipMemcpy(c_cublas, dc_cublas, size, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);

    free(a);
    free(b);
    free(c);
    free(c_cublas);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dc_cublas);

    return 0;
}
