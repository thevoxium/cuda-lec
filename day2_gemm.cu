
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <stdlib.h>

#define __threadCount 16

__global__ void gemmNaive(float* da, float* db, float* dc, int N){
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < N && j < N){
        float sum = 0.0f;
        for (int k=0; k < N; ++k){
            sum += da[i * N + k] * db[k * N + j];
        }
        dc[i * N + j] = sum;
    }
}

int main(){
    printf(" N      NaiveTime(ms)  NaiveGFLOPs   NaiveGFLOPs/s   cuBLASTime(ms)  cuBLASGFLOPs   cuBLASGFLOPs/s\n");
    for(int N = 512; N <= 8192; N += 512){
        size_t size = N * N * sizeof(float);
        float* a = (float*)malloc(size);
        float* b = (float*)malloc(size);
        float* c = (float*)malloc(size);
        float* c_cublas = (float*)malloc(size);

        for(int i=0;i<N;i++)
            for(int j=0;j<N;j++){
                a[i*N+j] = i+j;
                b[i*N+j] = 2*(i+j);
            }

        float *da, *db, *dc, *dc_cublas;
        hipMalloc((void**)&da, size);
        hipMalloc((void**)&db, size);
        hipMalloc((void**)&dc, size);
        hipMalloc((void**)&dc_cublas, size);

        hipMemcpy(da, a, size, hipMemcpyHostToDevice);
        hipMemcpy(db, b, size, hipMemcpyHostToDevice);

        dim3 tpb(__threadCount, __threadCount, 1);
        dim3 bpg((N+__threadCount-1)/__threadCount, (N+__threadCount-1)/__threadCount, 1);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        gemmNaive<<<bpg, tpb>>>(da, db, dc, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float naiveTimeMs = 0.0f;
        hipEventElapsedTime(&naiveTimeMs, start, stop);

        float naiveGFLOPs = 2.0f * N * N * N / 1e9f;
        float naiveGFLOPsPerSec = naiveGFLOPs / (naiveTimeMs / 1000.0f);

        hipblasHandle_t handle;
        hipblasCreate(&handle);
        float alpha = 1.0f;
        float beta = 0.0f;

        hipEventRecord(start);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, db, N, da, N, &beta, dc_cublas, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float cublasTimeMs = 0.0f;
        hipEventElapsedTime(&cublasTimeMs, start, stop);

        float cublasGFLOPs = 2.0f * N * N * N / 1e9f;
        float cublasGFLOPsPerSec = cublasGFLOPs / (cublasTimeMs / 1000.0f);

        printf("%4d     %10.3f     %10.2f     %13.2f     %13.3f     %12.2f     %15.2f\n",
               N, naiveTimeMs, naiveGFLOPs, naiveGFLOPsPerSec,
               cublasTimeMs, cublasGFLOPs, cublasGFLOPsPerSec);

        hipblasDestroy(handle);
        free(a); free(b); free(c); free(c_cublas);
        hipFree(da); hipFree(db); hipFree(dc); hipFree(dc_cublas);
        hipEventDestroy(start); hipEventDestroy(stop);
    }
    return 0;
}
